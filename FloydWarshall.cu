#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaCheck.cuh"

#include <iostream>

#include "utils.h"

using namespace std;

#define TILE_DIM 32

void runCpuFloydWarshall(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;
    hipEventRecord(start, 0);

    cout << "running the algorithm on CPU" << endl;
    for (int k = 0; k < numVertex; k++) {
        for (int i = 0; i < numVertex; i++) {
            for (int j = 0; j < numVertex; j++) {
                int itoj = i * numVertex + j;
                int itok = i * numVertex + k;
                int ktoj = k * numVertex + j;

                if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                    parent[itoj] = k;
                    distance[itoj] = distance[itok] + distance[ktoj];
                }
            }
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

__global__
void floydWarshallSuperNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        int itoj = i * numVertex + j;
        int itok = i * numVertex + k;
        int ktoj = k * numVertex + j;

        if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
            parent[itoj] = k;
            distance[itoj] = distance[itok] + distance[ktoj];
        }
    }
}

__global__
void floydWarshallNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex) {
        for (int j = 0; j < numVertex; j++) {
            int itoj = i * numVertex + j;
            int itok = i * numVertex + k;
            int ktoj = k * numVertex + j;

            if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                parent[itoj] = k;
                distance[itoj] = distance[itok] + distance[ktoj];
            }
        }
    }
}

__global__
void floydWarshallOptimized(int numVertex, int k, int* distance, int* parent) {//G will be the adjacency matrix, P will be path matrix
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < numVertex){
        int itoj = numVertex * i + j;
        int itok = numVertex * i + k;
        int ktoj = numVertex * k + j;

        __shared__ int dist_itok;
        if (threadIdx.x == 0){
            dist_itok = distance[itok];
        }
        __syncthreads();

        if (dist_itok != INF && distance[ktoj] != INF && distance[itoj] > dist_itok + distance[ktoj]) {
            distance[itoj] = dist_itok + distance[ktoj];
            parent[itoj] = k;
        }
    }
}

__global__
void floydWarshallTiledPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = primary_tile_number * blockDim.y + threadIdx.y;
    int j = primary_tile_number * blockDim.x + threadIdx.x;
    if(i<numVertex && j<numVertex){
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (i-tx+k <numVertex && j-ty+k<numVertex &&
                distance[itoj - tx + k] != INF && distance[itoj - ty * numVertex + k * numVertex] != INF &&
                distance[itoj] > distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex]) {

                distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex];
                parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
            __syncthreads();
        }
    }
}

__global__
void floydWarshallTiledPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i, j;

    // 1st row of blocks for row
    if (blockIdx.y == 0) {
        i = primary_tile_number * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (i-tx+k-blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex && j-ty+k < numVertex &&
                    distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                    distance[itoj - ty * numVertex + k * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x]
                    + distance[itoj - ty * numVertex + k * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex];
                    parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                __syncthreads();
            }
        }
    }

    // 2nd row of blocks for columns
    if (blockIdx.y == 1) {
        i = blockIdx.x * blockDim.y + threadIdx.y;
        j = primary_tile_number * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (i-tx+k < numVertex && j-(ty-k)- (blockIdx.x - primary_tile_number) * blockDim.x < numVertex &&
                    distance[itoj - tx + k] != INF &&
                    distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k]
                    + distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex];
                    parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                __syncthreads();
            }
        }
    }

    
}

__global__
void floydWarshallTiledPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (i-tx+k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex &&
                j-ty+k - (blockIdx.y - primary_tile_number) * blockDim.y < numVertex &&
                distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex] != INF &&
                distance[itoj] > distance[itoj - (tx - k) - (blockIdx.x - primary_tile_number) * blockDim.x]
                + distance[itoj - (ty - k) * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex]) {

                distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex];
                parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
        }
    }
}

__global__ void floydWarshallTiledSharedPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    __shared__ int s_distance[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = TILE_DIM * primary_tile_number + ty;
    int j = TILE_DIM * primary_tile_number + tx;

    int itoj = i * numVertex + j;
   
    s_distance[ty][tx] = distance[itoj];
    
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        __syncthreads();
        if (s_distance[ty][k] != INF &&
            s_distance[k][tx] != INF &&
            s_distance[ty][tx] > s_distance[ty][k] + s_distance[k][tx]) {

            s_distance[ty][tx] = s_distance[ty][k] + s_distance[k][tx];
        }
        __syncthreads();
    }
    distance[itoj] = s_distance[ty][tx];
}

__global__ void floydWarshallTiledSharedPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    __shared__ int s_distancePrimaryTile[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];

    int i = TILE_DIM * primary_tile_number + threadIdx.y;
    int j = TILE_DIM * primary_tile_number + threadIdx.x;
    
    int idxPrimaryTile = i * numVertex + j;
    s_distancePrimaryTile[threadIdx.y][threadIdx.x] = distance[idxPrimaryTile];
    __syncthreads();

    int idxCurrentTile;
    int shortestDistance;

    if (blockIdx.y == 0) { // 1st row of blocks for rows
        i = TILE_DIM * primary_tile_number + threadIdx.y;
        j = TILE_DIM * blockIdx.x + threadIdx.x;
        idxCurrentTile = i * numVertex + j;
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
        __syncthreads();

        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distancePrimaryTile[threadIdx.y][k] + s_distanceCurrentTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[threadIdx.y][k] != INF &&
                s_distanceCurrentTile[k][threadIdx.x] != INF &&
                newDistance < shortestDistance) {

                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    } else { // 2nd row of blocks for column
        i = TILE_DIM * blockIdx.x + threadIdx.y;
        j = TILE_DIM * primary_tile_number + threadIdx.x;
        idxCurrentTile = i * numVertex + j;
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
        __syncthreads();

        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distanceCurrentTile[threadIdx.y][k] + s_distancePrimaryTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[k][threadIdx.x] != INF &&
                s_distanceCurrentTile[threadIdx.y][k] != INF &&
                newDistance < shortestDistance) {

                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    }

    distance[idxCurrentTile] = shortestDistance;
}

__global__ void floydWarshallTiledSharedPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {

    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }

    __shared__ int s_distancePrimaryRow[TILE_DIM][TILE_DIM];
    __shared__ int s_distancePrimaryCol[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];
    
    int i, j;

    i = TILE_DIM * primary_tile_number + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    s_distancePrimaryRow[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * primary_tile_number + threadIdx.x;
    s_distancePrimaryCol[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    __syncthreads();

    int shortestDist = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        int newDistance = s_distancePrimaryCol[threadIdx.y][k] + s_distancePrimaryRow[k][threadIdx.x];
        if (s_distancePrimaryCol[threadIdx.y][k] != INF &&
            s_distancePrimaryRow[k][threadIdx.x] != INF &&
            newDistance < shortestDist) {

            shortestDist = newDistance;
        }
    }
     // __syncthreads();
    distance[i * numVertex + j] = shortestDist;
}

void runFloydWarshallSuperNaive(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallSuperNaive << <(numVertex - 1) / TILE_DIM + 1, (numVertex - 1) / TILE_DIM + 1 >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallNaive(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;   
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallNaive << <(numVertex - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallOptimized(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    dim3 dimGrid((numVertex - 1) / THREADS_PER_BLOCK + 1, numVertex);

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallOptimized << <dimGrid, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallTiled(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numDiagonalTiles; k++) {
        cout << "Phase number " << k << endl;
        floydWarshallTiledPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}


void runFloydWarshallTiledShared(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numDiagonalTiles; k++) {
        cout << "Phase number " << k << endl;
        floydWarshallTiledSharedPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledSharedPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledSharedPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

int main(int argc, char* argv[]) {

    if (argc < 3) {
        cout << "Please provide algorithm and input file as a command line argument" << endl;
        return 0;
    }
    string pathDataset("../data/");
    string algorithm(argv[1]);
    string pathGraphFile(pathDataset+string(argv[2]));

    int numVertex, numEdges;
    int* costMatrix = fileToCostMatrix(pathGraphFile, numVertex, numEdges);
        
    int* parent = (int*)malloc(numVertex * numVertex * sizeof(int));
    int* distance = (int*)malloc(numVertex * numVertex * sizeof(int));

    APSPInitDistanceParent(numVertex, costMatrix, distance, parent);

    if (algorithm == "0") {
        runCpuFloydWarshall(numVertex, distance, parent);
    } else if (algorithm == "1") {
        runFloydWarshallSuperNaive(numVertex, distance, parent);
    } else if (algorithm == "2") {
        runFloydWarshallNaive(numVertex, distance, parent);
    } else if (algorithm == "3") {
        runFloydWarshallOptimized(numVertex, distance, parent);
    } else if (algorithm == "4") {
        runFloydWarshallTiled(numVertex, distance, parent);
    } else if (algorithm == "2") {
        runFloydWarshallTiledShared(numVertex, distance, parent);
    }
    //  printPathAPSP(numVertex, distance, parent);
    string pathOutputFile(string("../output/fw") + algorithm + string(".txt"));
    // writeOutPathAPSP(pathOutputFile, numVertex, distance, parent);
}