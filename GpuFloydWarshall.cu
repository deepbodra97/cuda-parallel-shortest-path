#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaCheck.cuh"

#include <iostream>

#include "utils.h"

using namespace std;

#define TILE_DIM 3

void floydWarshall(int numVertex, int* distance, int* parent) {

    for (int k = 0; k < numVertex; k++) {
        for (int i = 0; i < numVertex; i++) {
            for (int j = 0; j < numVertex; j++) {
                int itoj = i * numVertex + j;
                int itok = i * numVertex + k;
                int ktoj = k * numVertex + j;

                if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                    parent[itoj] = k;
                    distance[itoj] = distance[itok] + distance[ktoj];
                }
            }
        }
    }
}

__global__
void floydWarshallNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex) {
        for (int j = 0; j < numVertex; j++) {
            int itoj = i * numVertex + j;
            int itok = i * numVertex + k;
            int ktoj = k * numVertex + j;

            if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                parent[itoj] = k;
                distance[itoj] = distance[itok] + distance[ktoj];
            }
        }
    }
}

__global__
void floydWarshallOptimized(int numVertex, int k, int* distance, int* parent) {//G will be the adjacency matrix, P will be path matrix
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < numVertex){
        int itoj = numVertex * i + j;
        int itok = numVertex * i + k;
        int ktoj = numVertex * k + j;

        __shared__ int dist_itok;
        if (threadIdx.x == 0){
            dist_itok = distance[itok];
        }
        __syncthreads();

        if (dist_itok != INF && distance[ktoj] != INF && distance[itoj] > dist_itok + distance[ktoj]) {
            distance[itoj] = dist_itok + distance[ktoj];
            parent[itoj] = k;
        }
    }
}

__global__
void floydWarshallTiledPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = primary_tile_number * blockDim.y + threadIdx.y;
    int j = primary_tile_number * blockDim.x + threadIdx.x;
    int itoj = i * numVertex + j;
    for (int k = 0; k < TILE_DIM; k++) {
        if (distance[itoj - tx + k] != INF && distance[itoj - ty * numVertex + k * numVertex] != INF &&
            distance[itoj] > distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex]) {

            distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex];
        }
        __syncthreads();
    }
}

__global__
void floydWarshallTiledPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i, j;

    // 1st row of blocks for columns
    if (blockIdx.y == 0 && blockIdx.x != primary_tile_number) {
        i = blockIdx.x * blockDim.y + threadIdx.y;
        j = primary_tile_number * blockDim.x + threadIdx.x;
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (distance[itoj - tx + k] != INF &&
                distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex] != INF &&
                distance[itoj] > distance[itoj - tx + k]
                + distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex]) {

                distance[itoj] = distance[itoj - tx  + k] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex];

            }
            __syncthreads();
        }
    }

    // 2nd row of blocks for rows
    if (blockIdx.y == 1 && blockIdx.x != primary_tile_number) {
        i = primary_tile_number * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                distance[itoj - ty * numVertex + k * numVertex] != INF &&
                distance[itoj] > distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x]
                + distance[itoj - ty * numVertex + k * numVertex]) {

                distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex];
            }
            __syncthreads();
        }
    }
}

__global__
void floydWarshallTiledPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int itoj = i * numVertex + j;
    if ((blockIdx.x != primary_tile_number) && (blockIdx.y != primary_tile_number)) { 
        for (int k = 0; k < TILE_DIM; k++) {
            if (distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex] != INF &&
                distance[itoj] > distance[itoj - (tx - k) - (blockIdx.x - primary_tile_number) * blockDim.x]
                + distance[itoj - (ty - k) * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex]) {
                
                distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex];
            }
        }
    }
}




void runFloydWarshallNaive(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    for (int k = 0; k < numVertex; k++) {
        floydWarshallNaive << <(numVertex - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
    }

    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

void runFloydWarshallOptimized(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    dim3 dimGrid((numVertex - 1) / THREADS_PER_BLOCK + 1, numVertex);

    for (int k = 0; k < numVertex; k++) {
        floydWarshallOptimized << <dimGrid, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
    }

    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

void runFloydWarshallTiled(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    for (int k = 0; k < numDiagonalTiles; k++) {
        floydWarshallTiledPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();

        
    }
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

int main() {
    int h_costMatrix[] = { 
        INF, 1, 5, INF, INF, INF,
        INF, INF, 2, 2, 1, INF,
        INF, INF, INF, INF, 2, INF,
        INF, INF, INF, INF, 3, 1,
        INF, INF, INF, INF, INF, 2,
        INF, INF, INF, INF, INF, INF,
    };

    int numVertex = 6;

    int* parent = (int*)malloc(numVertex * numVertex * sizeof(int));
    int* distance = (int*)malloc(numVertex * numVertex * sizeof(int));

    // fill(parent, parent + numVertex * numVertex, -1);
    // fill(distance, distance + numVertex * numVertex, INF);

    for (int i = 0; i < numVertex; i++) {
        for(int j = 0; j < numVertex; j++){
            if (i == j) {
                distance[i * numVertex + j] = 0;
                parent[i * numVertex + j] = -1;
            }
            else if (h_costMatrix[i * numVertex + j] == INF) {
                distance[i * numVertex + j] = INF;
                parent[i * numVertex + j] = -1;
            }
            else {
                distance[i * numVertex + j] = h_costMatrix[i * numVertex + j];
                parent[i * numVertex + j] = i;
            }
        }
    }

    // floydWarshall(numVertex, distance, parent);

    // runFloydWarshallNaive(numVertex, distance, parent);
    
    // runFloydWarshallOptimized(numVertex, distance, parent);

    runFloydWarshallTiled(numVertex, distance, parent);

    for (int i = 0; i < numVertex; i++) {
        for (int j = 0; j < numVertex; j++) {
            cout<<distance[i * numVertex + j] << " ";
        }
        cout << endl;
    }
}