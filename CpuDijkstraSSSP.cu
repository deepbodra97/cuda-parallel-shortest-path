#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "utils.h"

using namespace std;

int extractMin(int numVertex, int* distance, bool* visited) {
    int minNode = -1;
    int minDistance = INF;
    for (int i = 0; i < numVertex; i++) {
        if (!visited[i] && distance[i] < minDistance) {
            minDistance = distance[i];
            minNode = i;
        }
    }
    return minNode;
}

void dijkstra(int numVertex, int src, int *costMatrix, bool *visited, int * distance, int *parent) {
    distance[src] = 0;

    for (int i = 0; i < numVertex - 1; i++) {
        int u = extractMin(numVertex, distance, visited);
        if (u == -1) { // no min node to explore
            break;
        }
        visited[u] = true;
        for (int v = 0; v < numVertex; v++) {
            if (!visited[v] && costMatrix[u * numVertex + v] != INF && (distance[u] + costMatrix[u * numVertex + v]) < distance[v]){
                parent[v] = u;
                distance[v] = distance[u] + costMatrix[u * numVertex + v];
            }
        }
    }
}

void dijkstra(struct Graph* graph, int src, bool* visited, int* distance, int* parent) {
    distance[src] = 0;

    for (int i = 0; i < graph->numVertex - 1; i++) {
        int u = extractMin(graph->numVertex, distance, visited);
        if (u == -1) { // no min node to explore
            break;
        }
        visited[u] = true;
        struct AdjacencyListNode* neighbor = graph->neighbors[u].head;
        while(neighbor != NULL){
            if (!visited[neighbor->dest] && (distance[u] + neighbor->cost) < distance[neighbor->dest]) {
                parent[neighbor->dest] = u;
                distance[neighbor->dest] = distance[u] + neighbor->cost;
            }
            neighbor = neighbor->next;
        }
    }
}


//int main() {
//
//    /* Adjacency Matrix */
//    /*int numVertex = 3000;
//    int src = 1;
//
//    int* costMatrix = (int*) malloc(numVertex * numVertex * sizeof(int));
//    if (costMatrix == NULL) {
//        cout << "malloc failed" << endl;
//    }
//    fill(costMatrix, costMatrix + numVertex * numVertex, INF);
//
//    fileToCostMatrix(string("nyc-d.txt"), numVertex, costMatrix);
//
//    int* parent = (int*) malloc(numVertex * sizeof(int));
//    int* distance = (int*) malloc(numVertex * sizeof(int));
//    bool* visited = (bool*) malloc(numVertex * sizeof(bool));
//
//    fill(distance, distance + numVertex, INF);
//    fill(visited, visited + numVertex, false);
//    fill(parent, parent + numVertex, -1);
//
//    dijkstra(numVertex, src, (int*)costMatrix, visited, distance, parent);
//    printPathSSSP(numVertex, distance, parent);*/
//
//    /* Adjacency Linked List */
//    /*int numVertex = 264346;
//    int src = 1;
//
//    struct Graph* graph = (struct Graph*)malloc(sizeof(struct Graph));
//    
//    graph->numVertex = numVertex;
//    graph->neighbors = (struct AdjacencyList*) malloc(numVertex * sizeof(struct AdjacencyList));
//    
//    for (int i = 0; i < numVertex; ++i) {
//        graph->neighbors[i].head = NULL;
//    }
//
//    graph = fileToAdjacencyList(string("nyc-d.txt"), graph);
//
//
//    int* costMatrix = (int*)malloc(numVertex * numVertex * sizeof(int));
//    if (costMatrix == NULL) {
//        cout << "malloc failed" << endl;
//    }
//    fill(costMatrix, costMatrix + numVertex * numVertex, INF);
//
//    int* parent = (int*)malloc(numVertex * sizeof(int));
//    int* distance = (int*)malloc(numVertex * sizeof(int));
//    bool* visited = (bool*)malloc(numVertex * sizeof(bool));
//
//    fill(distance, distance + numVertex, INF);
//    fill(visited, visited + numVertex, false);
//    fill(parent, parent + numVertex, -1);
//
//    dijkstra(graph, src, visited, distance, parent);
//    printPathSSSP(numVertex, distance, parent);*/
//}