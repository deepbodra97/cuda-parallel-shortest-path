#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaCheck.cuh"

#include <iostream>

#include "utils.h"

using namespace std;

__global__
void bellmanFordRelax(int numVertex, int* vertices, int* indices, int* edges, int* weights, int* prev_distance, int* distance, int* parent) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertex) {
        for (int j = indices[tid]; j < indices[tid + 1]; j++) {
            int v = edges[j];
            int w = weights[j];

            if (prev_distance[tid] != INF && (prev_distance[tid] + w) < prev_distance[v]) {
                //parent[v] = i; // atomic
                atomicMin(&distance[v], prev_distance[tid] + w);
            }
        }
        /*if (prev_distance[tid] > distance[tid]) {
            prev_distance[tid] = distance[tid];
        }*/
        prev_distance[tid] = distance[tid];
    }
}

__global__
void bellmanFordParent(int numVertex, int* vertices, int* indices, int* edges, int* weights, int* distance, int* parent) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertex) {
        for (int j = indices[tid]; j < indices[tid + 1]; j++) {
            int v = edges[j];
            int w = weights[j];

            if (distance[tid] != INF && (distance[tid] + w) == distance[v]) {
                parent[v] = tid;
            }
        }
    }
}



//__global__
//void bellmanFordUpdateDistance() {
//
//}

int main() {

    ///* Adjacency Matrix */
    //int h_costMatrix[6][6] = {
    //    {INF, 1, 5, INF, INF, INF},
    //    {INF, INF, 2, 2, 1, INF},
    //    {INF, INF, INF, INF, 2, INF},
    //    {INF, INF, INF, INF, 3, 1},
    //    {INF, INF, INF, INF, INF, 2},
    //    {INF, INF, INF, INF, INF, INF},
    //};
    //int numVertex = 6;
    //int src = 1;

    //int* costMatrix = (int*)malloc(numVertex * numVertex * sizeof(int));
    //if (costMatrix == NULL) {
    //    cout << "malloc failed" << endl;
    //}
    //fill(costMatrix, costMatrix + numVertex * numVertex, INF);

    //for (int i = 0; i < numVertex; i++) {
    //    for (int j = 0; j < numVertex; j++) {
    //        costMatrix[i * numVertex + j] = h_costMatrix[i][j];
    //    }
    //}

    //// fileToCostMatrix(string("nyc-d.txt"), numVertex, costMatrix);

    //int* parent = (int*)malloc(numVertex * sizeof(int));
    //int* distance = (int*)malloc(numVertex * sizeof(int));

    //fill(distance, distance + numVertex, INF);
    //fill(parent, parent + numVertex, -1);

    //bellmanFord(numVertex, src, (int*)costMatrix, distance, parent);
    //printPathSSSP(numVertex, distance, parent);

    // int numVertex, numEdges;
    // map<int, list< pair<int, int > > > adjacencyList;

    // fileToAdjacencyList(string("nyc-d.txt"), adjacencyList, numVertex, numEdges);
    // cout << adjacencyList.size() << " " << numVertex << " " << numEdges << endl;
    // vector<int> vertices, indices, edges, weights;
    /*vertices.reserve(numVertex);
    indices.reserve(numVertex + 1);
    edges.reserve(numEdges);
    weights.reserve(numEdges);*/
    // adjacencyListToCSR(adjacencyList, vertices, indices, edges, weights);
    /*for (auto i : weights)
        std::cout << i << ' ';*/

     int numVertex = 6, numEdges = 9;
     vector<int> vertices = { 0, 1, 2, 3, 4, 5 }, indices = { 0, 2, 5, 6, 8, 9 }, edges = { 1, 2, 2, 3, 4, 4, 4, 5, 5 }, weights = { 1,5,2,2,1,2,3,1,2 };
    /*int numVertex, numEdges;
    vector<int> vertices, indices, edges, weights;
    map<int, list< pair<int, int > > > adjacencyList;    
    fileToAdjacencyList(string("nyc-d.txt"), adjacencyList, numVertex, numEdges);
    adjacencyListToCSR(adjacencyList, vertices, indices, edges, weights);*/

    int src = 0;

    int* parent = (int*)malloc(numVertex * sizeof(int));
    int* prev_distance = (int*)malloc(numVertex * sizeof(int));
    int* distance = (int*)malloc(numVertex * sizeof(int));

    
    fill(prev_distance, prev_distance + numVertex, INF);
    fill(distance, distance + numVertex, INF);
    fill(parent, parent + numVertex, -1);

    prev_distance[src] = 0;
    distance[src] = 0;

    int* d_vertices;
    int* d_indices;
    int* d_edges;
    int* d_weights;
    int* d_prev_distance;
    int* d_distance;
    int* d_parent;
    
    cudaCheck(hipMalloc((void**)&d_vertices, numVertex*sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_indices, (numVertex+1) * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_edges, numEdges * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_weights, numEdges * sizeof(int)));

    cudaCheck(hipMalloc((void**)&d_prev_distance, numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_vertices, vertices.data(), numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_indices, indices.data(), (numVertex+1) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_edges, edges.data(), numEdges * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weights, weights.data(), numEdges * sizeof(int), hipMemcpyHostToDevice));

    cudaCheck(hipMemcpy(d_prev_distance, prev_distance, numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * sizeof(int), hipMemcpyHostToDevice));

    for (int k = 0; k < numVertex - 1; k++) {
        bellmanFordRelax<<<(numVertex-1)/THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(numVertex, d_vertices, d_indices, d_edges, d_weights, d_prev_distance, d_distance, d_parent);
    }
    bellmanFordParent << <(numVertex - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (numVertex, d_vertices, d_indices, d_edges, d_weights, d_distance, d_parent);

    cudaCheck(hipMemcpy(distance, d_distance, numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * sizeof(int), hipMemcpyDeviceToHost));

    printPathSSSP(numVertex, distance, parent);
}