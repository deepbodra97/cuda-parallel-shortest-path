#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaCheck.cuh"

#include <iostream>

#include "utils.h"

using namespace std;

#define TILE_DIM 32

void runCpuFloydWarshall(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;
    hipEventRecord(start, 0);

    cout << "running the algorithm on CPU" << endl;
    for (int k = 0; k < numVertex; k++) {
        for (int i = 0; i < numVertex; i++) {
            for (int j = 0; j < numVertex; j++) {
                int itoj = i * numVertex + j;
                int itok = i * numVertex + k;
                int ktoj = k * numVertex + j;

                if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                    parent[itoj] = k;
                    distance[itoj] = distance[itok] + distance[ktoj];
                }
            }
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

__global__
void floydWarshallSuperNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        int itoj = i * numVertex + j;
        int itok = i * numVertex + k;
        int ktoj = k * numVertex + j;

        if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
            parent[itoj] = k;
            distance[itoj] = distance[itok] + distance[ktoj];
        }
    }
}

__global__
void floydWarshallNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex) {
        for (int j = 0; j < numVertex; j++) {
            int itoj = i * numVertex + j;
            int itok = i * numVertex + k;
            int ktoj = k * numVertex + j;

            if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                parent[itoj] = k;
                distance[itoj] = distance[itok] + distance[ktoj];
            }
        }
    }
}

__global__
void floydWarshallOptimized(int numVertex, int k, int* distance, int* parent) {//G will be the adjacency matrix, P will be path matrix
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < numVertex){
        int itoj = numVertex * i + j;
        int itok = numVertex * i + k;
        int ktoj = numVertex * k + j;

        __shared__ int dist_itok;
        if (threadIdx.x == 0){
            dist_itok = distance[itok];
        }
        __syncthreads();

        if (dist_itok != INF && distance[ktoj] != INF && distance[itoj] > dist_itok + distance[ktoj]) {
            distance[itoj] = dist_itok + distance[ktoj];
            parent[itoj] = k;
        }
    }
}

__global__
void floydWarshallTiledPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = primary_tile_number * blockDim.y + threadIdx.y;
    int j = primary_tile_number * blockDim.x + threadIdx.x;
    if(i<numVertex && j<numVertex){
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (j-tx+k <numVertex && i-ty+k<numVertex && // i-ty?
                distance[itoj - tx + k] != INF && distance[itoj - ty * numVertex + k * numVertex] != INF &&
                distance[itoj] > distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex]) {

                distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex];
                // parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
            // __syncthreads();
        }
    }
}

__global__
void floydWarshallTiledPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i, j;

    // 1st row of blocks for row
    if (blockIdx.y == 0) {
        i = primary_tile_number * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (j-tx+k-blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex && i-ty+k < numVertex &&
                    distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                    distance[itoj - ty * numVertex + k * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x]
                    + distance[itoj - ty * numVertex + k * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex];
                    // parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                // __syncthreads();
            }
        }
    }

    // 2nd row of blocks for columns
    if (blockIdx.y == 1) {
        i = blockIdx.x * blockDim.y + threadIdx.y;
        j = primary_tile_number * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (j-tx+k < numVertex && i-(ty-k)- (blockIdx.x - primary_tile_number) * blockDim.x < numVertex &&
                    distance[itoj - tx + k] != INF &&
                    distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k]
                    + distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex];
                    // parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                // __syncthreads();
            }
        }
    }
}

__global__
void floydWarshallTiledPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (j-tx+k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex &&
                i-ty+k - (blockIdx.y - primary_tile_number) * blockDim.y < numVertex &&
                distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex] != INF &&
                distance[itoj] > distance[itoj - (tx - k) - (blockIdx.x - primary_tile_number) * blockDim.x]
                + distance[itoj - (ty - k) * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex]) {

                distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex];
                // parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
        }
    }
}

__global__ void floydWarshallTiledSharedPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    __shared__ int s_distance[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = TILE_DIM * primary_tile_number + ty;
    int j = TILE_DIM * primary_tile_number + tx;
    int itoj = i * numVertex + j;

    int shortestParent;
    if (i < numVertex && j < numVertex) {
        s_distance[ty][tx] = distance[itoj];
        shortestParent = parent[itoj];
    } else {
        s_distance[ty][tx] = INF;
        shortestParent = -1;
    }
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        __syncthreads();
        if (s_distance[ty][k] != INF &&
            s_distance[k][tx] != INF &&
            s_distance[ty][tx] > s_distance[ty][k] + s_distance[k][tx]) {

            s_distance[ty][tx] = s_distance[ty][k] + s_distance[k][tx];
            shortestParent = TILE_DIM * primary_tile_number + k;
        }
        __syncthreads();
    }
    if (i < numVertex && j < numVertex) {
        distance[itoj] = s_distance[ty][tx];
        parent[itoj] = shortestParent;
    }
}

__global__ void floydWarshallTiledSharedPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    __shared__ int s_distancePrimaryTile[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];

    int i = TILE_DIM * primary_tile_number + threadIdx.y;
    int j = TILE_DIM * primary_tile_number + threadIdx.x;
    
    int idxPrimaryTile = i * numVertex + j;

    if (i < numVertex && j < numVertex) {
        s_distancePrimaryTile[threadIdx.y][threadIdx.x] = distance[idxPrimaryTile];
    }
    else {
        s_distancePrimaryTile[threadIdx.y][threadIdx.x] = INF;
    }
    __syncthreads();

    int idxCurrentTile;
    int shortestDistance;
    int shortestParent;

    if (blockIdx.y == 0) { // 1st row of blocks for rows
        i = TILE_DIM * primary_tile_number + threadIdx.y;
        j = TILE_DIM * blockIdx.x + threadIdx.x;
        idxCurrentTile = i * numVertex + j;

        if (i < numVertex && j < numVertex) {
            s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
            shortestParent = parent[idxCurrentTile];
        }
        else {
            s_distanceCurrentTile[threadIdx.y][threadIdx.x] = INF;
            shortestParent = -1;
        }
        __syncthreads();

        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

        #pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distancePrimaryTile[threadIdx.y][k] + s_distanceCurrentTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[threadIdx.y][k] != INF &&
                s_distanceCurrentTile[k][threadIdx.x] != INF &&
                newDistance < shortestDistance) {

                shortestParent = TILE_DIM * primary_tile_number + k;
                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    } else { // 2nd row of blocks for column
        i = TILE_DIM * blockIdx.x + threadIdx.y;
        j = TILE_DIM * primary_tile_number + threadIdx.x;
        idxCurrentTile = i * numVertex + j;
        
        if (i < numVertex && j < numVertex) {
            s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
            shortestParent = parent[idxCurrentTile];
        }
        else {
            s_distanceCurrentTile[threadIdx.y][threadIdx.x] = INF;
            shortestParent = -1;
        }
        __syncthreads();
        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

        #pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distanceCurrentTile[threadIdx.y][k] + s_distancePrimaryTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[k][threadIdx.x] != INF &&
                s_distanceCurrentTile[threadIdx.y][k] != INF &&
                newDistance < shortestDistance) {

                shortestParent = TILE_DIM * primary_tile_number + k;
                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    }
    if (i < numVertex && j < numVertex) {
        distance[idxCurrentTile] = shortestDistance;
        parent[idxCurrentTile] = shortestParent;
    }
}

__global__ void floydWarshallTiledSharedPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {

    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }

    __shared__ int s_distancePrimaryRow[TILE_DIM][TILE_DIM];
    __shared__ int s_distancePrimaryCol[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];
    
    int i, j;

    i = TILE_DIM * primary_tile_number + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        s_distancePrimaryRow[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];
    }
    else {
        s_distancePrimaryRow[threadIdx.y][threadIdx.x] = INF;
    }
    

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * primary_tile_number + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        s_distancePrimaryCol[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];
    }
    else {
        s_distancePrimaryCol[threadIdx.y][threadIdx.x] = INF;
    }

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    int shortestParent;
    if (i < numVertex && j < numVertex) {
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];
        shortestParent = parent[i * numVertex + j];
    }
    else {
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = INF;
        shortestParent = -1;
    }

    __syncthreads();

    int shortestDist = s_distanceCurrentTile[threadIdx.y][threadIdx.x];
    #pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        int newDistance = s_distancePrimaryCol[threadIdx.y][k] + s_distancePrimaryRow[k][threadIdx.x];
        if (s_distancePrimaryCol[threadIdx.y][k] != INF &&
            s_distancePrimaryRow[k][threadIdx.x] != INF &&
            newDistance < shortestDist) {
            shortestParent = TILE_DIM * primary_tile_number + k;
            shortestDist = newDistance;
        }
    }
     // __syncthreads();
    if(i<numVertex && j<numVertex){
        distance[i * numVertex + j] = shortestDist;
        parent[i * numVertex + j] = shortestParent;
    }
}

void runFloydWarshallSuperNaive(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallSuperNaive << <(numVertex - 1) / TILE_DIM + 1, (numVertex - 1) / TILE_DIM + 1 >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallNaive(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;   
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallNaive << <(numVertex - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallOptimized(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    dim3 dimGrid((numVertex - 1) / THREADS_PER_BLOCK + 1, numVertex);

    // run kernel
    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numVertex; k++) {
        floydWarshallOptimized << <dimGrid, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

void runFloydWarshallTiled(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numDiagonalTiles; k++) {
        floydWarshallTiledPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
        floydWarshallTiledPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
        floydWarshallTiledPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}


void runFloydWarshallTiledShared(int numVertex, int* distance, int* parent) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;

    hipEventRecord(start, 0);

    int* d_distance;
    int* d_parent;

    // allocate memory on GPU and copy data from CPU to GPU
    cout << "allocating data on GPU" << endl;
    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cout << "copying data to GPU" << endl;
    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    cout << "Kernel is executing" << endl;
    for (int k = 0; k < numDiagonalTiles; k++) {
        floydWarshallTiledSharedPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
        floydWarshallTiledSharedPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
        floydWarshallTiledSharedPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
    }

    // copy results to CPU
    cout << "copying results to CPU" << endl;
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "Time: " << duration << "ms" << endl;
}

int main(int argc, char* argv[]) {

    if (argc < 5) {
        cout << "Please provide proper command line arguments" << endl;
        return 0;
    }
    string pathDataset("../data/");
    string algorithm(argv[1]);
    string pathGraphFile(pathDataset+string(argv[2]));
    string validate(argv[3]);
    string outputFormat(argv[4]);

    int numVertex, numEdges;
    int* costMatrix = fileToCostMatrix(pathGraphFile, numVertex, numEdges);
        
    int* parent = (int*)malloc(numVertex * numVertex * sizeof(int));
    int* distance = (int*)malloc(numVertex * numVertex * sizeof(int));

    APSPInitDistanceParent(numVertex, costMatrix, distance, parent);

    if (algorithm == "0") {
        runCpuFloydWarshall(numVertex, distance, parent);
    } else{
        if (algorithm == "1") {
            runFloydWarshallSuperNaive(numVertex, distance, parent);
        } else if (algorithm == "2") {
            runFloydWarshallNaive(numVertex, distance, parent);
        } else if (algorithm == "3") {
            runFloydWarshallOptimized(numVertex, distance, parent);
        } else if (algorithm == "4") {
            runFloydWarshallTiled(numVertex, distance, parent);
        } else if (algorithm == "5") {
            runFloydWarshallTiledShared(numVertex, distance, parent);
        }

        if (validate == "true") {
            int* exp_parent = (int*)malloc(numVertex * sizeof(int));
            int* exp_distance = (int*)malloc(numVertex * sizeof(int));
            APSPInitDistanceParent(numVertex, costMatrix, distance, parent);
            runCpuFloydWarshall(numVertex, distance, parent);
            validateDistanceAPSP(numVertex, exp_distance, distance);
        }
    }
    //  
    if (outputFormat == "print") {
        printPathAPSP(numVertex, distance, parent);
    } else if (outputFormat == "write") {
        string pathOutputFile(string("../output/fw") + algorithm + string(".txt"));
        writeOutPathAPSP(pathOutputFile, numVertex, distance, parent);
    } else if (outputFormat == "none") {

    } else {
        cout << "Illegal output format argument" << endl;
    }
}