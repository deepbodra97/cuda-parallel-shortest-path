#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaCheck.cuh"

#include <iostream>

#include "utils.h"

using namespace std;

#define TILE_DIM 32

void floydWarshall(int numVertex, int* distance, int* parent) {

    for (int k = 0; k < numVertex; k++) {
        for (int i = 0; i < numVertex; i++) {
            for (int j = 0; j < numVertex; j++) {
                int itoj = i * numVertex + j;
                int itok = i * numVertex + k;
                int ktoj = k * numVertex + j;

                if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                    parent[itoj] = k;
                    distance[itoj] = distance[itok] + distance[ktoj];
                }
            }
        }
    }
}

__global__
void floydWarshallNaive(int numVertex, int k, int* distance, int* parent) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex) {
        for (int j = 0; j < numVertex; j++) {
            int itoj = i * numVertex + j;
            int itok = i * numVertex + k;
            int ktoj = k * numVertex + j;

            if (distance[itok] != INF && distance[ktoj] != INF && distance[itoj] > distance[itok] + distance[ktoj]) {
                parent[itoj] = k;
                distance[itoj] = distance[itok] + distance[ktoj];
            }
        }
    }
}

__global__
void floydWarshallOptimized(int numVertex, int k, int* distance, int* parent) {//G will be the adjacency matrix, P will be path matrix
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < numVertex){
        int itoj = numVertex * i + j;
        int itok = numVertex * i + k;
        int ktoj = numVertex * k + j;

        __shared__ int dist_itok;
        if (threadIdx.x == 0){
            dist_itok = distance[itok];
        }
        __syncthreads();

        if (dist_itok != INF && distance[ktoj] != INF && distance[itoj] > dist_itok + distance[ktoj]) {
            distance[itoj] = dist_itok + distance[ktoj];
            parent[itoj] = k;
        }
    }
}

__global__
void floydWarshallTiledPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = primary_tile_number * blockDim.y + threadIdx.y;
    int j = primary_tile_number * blockDim.x + threadIdx.x;
    if(i<numVertex && j<numVertex){
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (i-tx+k <numVertex && j-ty+k<numVertex &&
                distance[itoj - tx + k] != INF && distance[itoj - ty * numVertex + k * numVertex] != INF &&
                distance[itoj] > distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex]) {

                distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex];
                parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
            __syncthreads();
        }
    }
}

__global__
void floydWarshallTiledPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i, j;

    // 1st row of blocks for row
    if (blockIdx.y == 0) {
        i = primary_tile_number * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (i-tx+k-blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex && j-ty+k < numVertex &&
                    distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                    distance[itoj - ty * numVertex + k * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x]
                    + distance[itoj - ty * numVertex + k * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex];
                    parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                __syncthreads();
            }
        }
    }

    // 2nd row of blocks for columns
    if (blockIdx.y == 1) {
        i = blockIdx.x * blockDim.y + threadIdx.y;
        j = primary_tile_number * blockDim.x + threadIdx.x;
        if (i < numVertex && j < numVertex) {
            int itoj = i * numVertex + j;
            for (int k = 0; k < TILE_DIM; k++) {
                if (i-tx+k < numVertex && j-(ty-k)- (blockIdx.x - primary_tile_number) * blockDim.x < numVertex &&
                    distance[itoj - tx + k] != INF &&
                    distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex] != INF &&
                    distance[itoj] > distance[itoj - tx + k]
                    + distance[itoj - (ty - k) * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex]) {

                    distance[itoj] = distance[itoj - tx + k] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.x - primary_tile_number) * blockDim.x * numVertex];
                    parent[itoj] = TILE_DIM * primary_tile_number + k;
                }
                __syncthreads();
            }
        }
    }

    
}

__global__
void floydWarshallTiledPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVertex && j < numVertex) {
        int itoj = i * numVertex + j;
        for (int k = 0; k < TILE_DIM; k++) {
            if (i-tx+k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x < numVertex &&
                j-ty+k - (blockIdx.y - primary_tile_number) * blockDim.y < numVertex &&
                distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] != INF &&
                distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex] != INF &&
                distance[itoj] > distance[itoj - (tx - k) - (blockIdx.x - primary_tile_number) * blockDim.x]
                + distance[itoj - (ty - k) * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex]) {

                distance[itoj] = distance[itoj - tx + k - blockIdx.x * blockDim.x + primary_tile_number * blockDim.x] + distance[itoj - ty * numVertex + k * numVertex - (blockIdx.y - primary_tile_number) * blockDim.y * numVertex];
                parent[itoj] = TILE_DIM * primary_tile_number + k;
            }
        }
    }
}

__global__ void floydWarshallTiledSharedPhase1(int numVertex, int primary_tile_number, int* distance, int* parent) {
    __shared__ int s_distance[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = TILE_DIM * primary_tile_number + ty;
    int j = TILE_DIM * primary_tile_number + tx;

    int itoj = i * numVertex + j;
   
    s_distance[ty][tx] = distance[itoj];
    
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        __syncthreads();
        if (s_distance[ty][k] != INF &&
            s_distance[k][tx] != INF &&
            s_distance[ty][tx] > s_distance[ty][k] + s_distance[k][tx]) {

            s_distance[ty][tx] = s_distance[ty][k] + s_distance[k][tx];
        }
        __syncthreads();
    }
    distance[itoj] = s_distance[ty][tx];
}

__global__ void floydWarshallTiledSharedPhase2(int numVertex, int primary_tile_number, int* distance, int* parent) {
    if (blockIdx.x == primary_tile_number) {
        return;
    }
    __shared__ int s_distancePrimaryTile[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];

    int i = TILE_DIM * primary_tile_number + threadIdx.y;
    int j = TILE_DIM * primary_tile_number + threadIdx.x;
    
    int idxPrimaryTile = i * numVertex + j;
    s_distancePrimaryTile[threadIdx.y][threadIdx.x] = distance[idxPrimaryTile];
    __syncthreads();

    int idxCurrentTile;
    int shortestDistance;

    if (blockIdx.y == 0) { // 1st row of blocks for rows
        i = TILE_DIM * primary_tile_number + threadIdx.y;
        j = TILE_DIM * blockIdx.x + threadIdx.x;
        idxCurrentTile = i * numVertex + j;
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
        __syncthreads();

        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distancePrimaryTile[threadIdx.y][k] + s_distanceCurrentTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[threadIdx.y][k] != INF &&
                s_distanceCurrentTile[k][threadIdx.x] != INF &&
                newDistance < shortestDistance) {

                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    } else { // 2nd row of blocks for column
        i = TILE_DIM * blockIdx.x + threadIdx.y;
        j = TILE_DIM * primary_tile_number + threadIdx.x;
        idxCurrentTile = i * numVertex + j;
        s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[idxCurrentTile];
        __syncthreads();

        shortestDistance = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            int newDistance = s_distanceCurrentTile[threadIdx.y][k] + s_distancePrimaryTile[k][threadIdx.x];
            // __syncthreads();
            if (s_distancePrimaryTile[k][threadIdx.x] != INF &&
                s_distanceCurrentTile[threadIdx.y][k] != INF &&
                newDistance < shortestDistance) {

                shortestDistance = newDistance;
            }
            __syncthreads();
        }
    }

    distance[idxCurrentTile] = shortestDistance;
}

__global__ void floydWarshallTiledSharedPhase3(int numVertex, int primary_tile_number, int* distance, int* parent) {

    if (blockIdx.x == primary_tile_number || blockIdx.y == primary_tile_number) {
        return;
    }

    __shared__ int s_distancePrimaryRow[TILE_DIM][TILE_DIM];
    __shared__ int s_distancePrimaryCol[TILE_DIM][TILE_DIM];
    __shared__ int s_distanceCurrentTile[TILE_DIM][TILE_DIM];
    
    int i, j;

    i = TILE_DIM * primary_tile_number + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    s_distancePrimaryRow[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * primary_tile_number + threadIdx.x;
    s_distancePrimaryCol[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    i = TILE_DIM * blockIdx.y + threadIdx.y;
    j = TILE_DIM * blockIdx.x + threadIdx.x;
    s_distanceCurrentTile[threadIdx.y][threadIdx.x] = distance[i * numVertex + j];

    __syncthreads();

    int shortestDist = s_distanceCurrentTile[threadIdx.y][threadIdx.x];

#pragma unroll
    for (int k = 0; k < TILE_DIM; k++) {
        int newDistance = s_distancePrimaryCol[threadIdx.y][k] + s_distancePrimaryRow[k][threadIdx.x];
        if (s_distancePrimaryCol[threadIdx.y][k] != INF &&
            s_distancePrimaryRow[k][threadIdx.x] != INF &&
            newDistance < shortestDist) {

            shortestDist = newDistance;
        }
    }
     // __syncthreads();
    distance[i * numVertex + j] = shortestDist;
}


void runFloydWarshallNaive(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    for (int k = 0; k < numVertex; k++) {
        floydWarshallNaive << <(numVertex - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }

    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

void runFloydWarshallOptimized(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    dim3 dimGrid((numVertex - 1) / THREADS_PER_BLOCK + 1, numVertex);

    for (int k = 0; k < numVertex; k++) {
        floydWarshallOptimized << <dimGrid, THREADS_PER_BLOCK >> > (numVertex, k, d_distance, d_parent);
    }

    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

void runFloydWarshallTiled(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    for (int k = 0; k < numDiagonalTiles; k++) {
        cout << "Phase number " << k << endl;
        floydWarshallTiledPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}


void runFloydWarshallTiledShared(int numVertex, int* distance, int* parent) {
    int* d_distance;
    int* d_parent;

    cudaCheck(hipMalloc((void**)&d_distance, numVertex * numVertex * sizeof(int)));
    cudaCheck(hipMalloc((void**)&d_parent, numVertex * numVertex * sizeof(int)));

    cudaCheck(hipMemcpy(d_distance, distance, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, parent, numVertex * numVertex * sizeof(int), hipMemcpyHostToDevice));

    int numDiagonalTiles = (numVertex - 1) / TILE_DIM + 1;

    dim3 dimGridPhase1(1, 1), dimGridPhase2(numDiagonalTiles, 2), dimGridPhase3(numDiagonalTiles, numDiagonalTiles);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    for (int k = 0; k < numDiagonalTiles; k++) {
        cout << "Phase number " << k << endl;
        floydWarshallTiledSharedPhase1 << <  dimGridPhase1, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledSharedPhase2 << <  dimGridPhase2, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
        floydWarshallTiledSharedPhase3 << <  dimGridPhase3, dimBlock >> > (numVertex, k, d_distance, d_parent);
        hipDeviceSynchronize();
    }
    cudaCheck(hipMemcpy(distance, d_distance, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(parent, d_parent, numVertex * numVertex * sizeof(int), hipMemcpyDeviceToHost));
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float duration;
    /*int h_costMatrix[] = { 
        INF, 1, 5, INF, INF, INF,
        INF, INF, 2, 2, 1, INF,
        INF, INF, INF, INF, 2, INF,
        INF, INF, INF, INF, 3, 1,
        INF, INF, INF, INF, INF, 2,
        INF, INF, INF, INF, INF, INF,
    };

    int numVertex = 6;*/

     int numVertex, numEdges;
     int* h_costMatrix = fileToCostMatrix(string("../data/gnutella25.txt"), numVertex, numEdges);
        
    int* parent = (int*)malloc(numVertex * numVertex * sizeof(int));
    int* distance = (int*)malloc(numVertex * numVertex * sizeof(int));

    // fill(parent, parent + numVertex * numVertex, -1);
    // fill(distance, distance + numVertex * numVertex, INF);

    for (int i = 0; i < numVertex; i++) {
        for(int j = 0; j < numVertex; j++){
            if (i == j) {
                distance[i * numVertex + j] = 0;
                parent[i * numVertex + j] = -1;
            }
            else if (h_costMatrix[i * numVertex + j] == INF) {
                distance[i * numVertex + j] = INF;
                parent[i * numVertex + j] = -1;
            }
            else {
                distance[i * numVertex + j] = h_costMatrix[i * numVertex + j];
                parent[i * numVertex + j] = i;
            }
        }
    }

    // floydWarshall(numVertex, distance, parent);
    // runFloydWarshallNaive(numVertex, distance, parent);
    // runFloydWarshallOptimized(numVertex, distance, parent);

    hipEventRecord(start, 0);
    runFloydWarshallTiled(numVertex, distance, parent);
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout << "tiled floyd warshall " << duration << "ms" << endl;

    /*hipEventRecord(start, 0);
    runFloydWarshallTiledShared(numVertex, distance, parent);
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    cout<<"tiled floyd warshall with shared memory "<< duration << "ms" << endl;*/

    /*for (int i = 0; i < numVertex; i++) {
        for (int j = 0; j < numVertex; j++) {
            cout<<distance[i * numVertex + j] << " ";
        }
        cout << endl;
    }*/

    printPathAPSP(numVertex, distance, parent);
    // writeOutPathAPSP("../output/path.txt", numVertex, distance, parent);
}