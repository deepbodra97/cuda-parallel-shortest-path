#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "cudaCheck.cuh"

#include "utils.h"

using namespace std;

#define THREADS_PER_BLOCK 512

__device__
int extractMin(int numVertex, int* distance, bool* visited, int src) {
    int minNode = -1;
    int minDistance = INF;
    for (int i = 0; i < numVertex; i++) {
        if (!visited[src * numVertex + i] && distance[src * numVertex + i] < minDistance) {
            minDistance = distance[src * numVertex + i];
            minNode = i;
        }
    }
    return minNode;
}

__global__
void dijkstra(int numVertex, int* costMatrix, bool* visited, int* distance, int* parent) {
    int src = blockIdx.x * blockDim.x + threadIdx.x;

    if (src < numVertex) {
        distance[src * numVertex + src] = 0;
        parent[src * numVertex + src] = -1;

        for (int i = 0; i < numVertex - 1; i++) {
            int u = extractMin(numVertex, distance, visited, src);
            if (u == -1) { // no min node to explore
                break;
            }
            visited[src * numVertex + u] = true;
            for (int v = 0; v < numVertex; v++) {
                if (!visited[src * numVertex + v] && costMatrix[u * numVertex + v] != INF && (distance[src * numVertex + u] + costMatrix[u * numVertex + v]) < distance[src * numVertex + v]){
                    parent[src * numVertex + v] = u;
                    distance[src * numVertex + v] = distance[src * numVertex + u] + costMatrix[u * numVertex + v];
                }
            }
        }
    }
}


int main() {
    int h_numVertex = 6;
    int h_costMatrix[6][6] = {
        {INF, 1, 5, INF, INF, INF},
        {INF, INF, 2, 2, 1, INF},
        {INF, INF, INF, INF, 2, INF},
        {INF, INF, INF, INF, 3, 1},
        {INF, INF, INF, INF, INF, 2},
        {INF, INF, INF, INF, INF, INF},
    };

    int* h_parent = (int*)malloc(h_numVertex * h_numVertex * sizeof(int));
    int* h_distance = (int*)malloc(h_numVertex * h_numVertex * sizeof(int));
    bool* h_visited = (bool*)malloc(h_numVertex * h_numVertex * sizeof(bool));

    fill(h_parent, h_parent + h_numVertex * h_numVertex, -1);
    fill(h_distance, h_distance + h_numVertex * h_numVertex, INF);
    fill(h_visited, h_visited + h_numVertex * h_numVertex, false);

    const int bytesNumVertex = sizeof(int);
    const int bytesCostMatrix = h_numVertex * h_numVertex * sizeof(int);

    int* d_costMatrix;
    int* d_parent;
    int* d_distance;
    bool* d_visited;

    cudaCheck(hipMalloc((void**)&d_costMatrix, bytesCostMatrix));
    cudaCheck(hipMalloc((void**)&d_parent, bytesCostMatrix));
    cudaCheck(hipMalloc((void**)&d_distance, bytesCostMatrix));
    cudaCheck(hipMalloc((void**)&d_visited, bytesCostMatrix));

    cudaCheck(hipMemcpy(d_costMatrix, h_costMatrix, bytesCostMatrix, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_parent, h_parent, bytesCostMatrix, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_distance, h_distance, bytesCostMatrix, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_visited, h_visited, bytesCostMatrix, hipMemcpyHostToDevice));

    dijkstra<<<(h_numVertex-1)/THREADS_PER_BLOCK+1, THREADS_PER_BLOCK>>>(h_numVertex, d_costMatrix, d_visited, d_distance, d_parent);

    cudaCheck(hipMemcpy(h_distance, d_distance, bytesCostMatrix, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(h_parent, d_parent, bytesCostMatrix, hipMemcpyDeviceToHost));
    
    printPathAPSP(h_numVertex, h_distance, h_parent);
}